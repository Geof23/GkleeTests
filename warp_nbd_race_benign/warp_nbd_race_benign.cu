
#include <hip/hip_runtime.h>
#define N 16

__global__ void k(int* in)
{
  if(threadIdx.x < N)
    in[0] = 0;
}

int main()
{
  int* din;
  hipMalloc((void**) &din, N*sizeof(int));
  k<<<1,N>>>(din);
}