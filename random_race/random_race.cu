
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

__global__ void device_global(unsigned int *input_array, int num_elements) {
  // This may race depending on the call to rand() when filling the array
  int my_index = blockIdx.x * blockDim.x + threadIdx.x;
  int index = input_array[my_index];
  input_array[index] = my_index;
}

int main(void) {
  // how big our array for interfacing with the GPU will be
  int num_elements = 32;
  int num_bytes = sizeof(unsigned int) * num_elements;
    
  // pointers for the interfacing arrays
  unsigned int *host_array = 0;
  unsigned int *device_array = 0;
 
  // malloc for host and device
  host_array = (unsigned int*) malloc(num_bytes);
  hipMalloc((void **) &device_array, num_bytes);

  // check the mallocs
  if (host_array == 0) {
    printf("Unable to allocate memory on host");
    return 1;
  }

  if (device_array == 0) {
    printf("Unable to allocate memory on device");
    return 1;
  }

  srand(time(NULL));
  
  // set host array values
  for (int i = 0; i<num_elements; i++) {
    host_array[i] = i;
    // At a very low rate cause the contents to be wrong: causing a race later
    if (rand() % 1000 == 3) {
      host_array[i] = (i-1)%32;
    }
  }

  // copy them to the GPU
  hipMemcpy(device_array, host_array, num_bytes, hipMemcpyHostToDevice);

  // define block and grid sizes
  int block_size = 32;
  int grid_size = (num_elements + block_size - 1) / block_size;

  // run GPU code
  device_global<<<grid_size, block_size>>>(device_array, num_elements);

  // copy output to host
  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  // print any information
  for (int i=0; i<num_elements; i++) {
    printf("%d, ", host_array[i]);
    if (i%16 == 15) {
      printf("\n");
    }
  }   
  

  // free memory
  free(host_array);
  hipFree(device_array);
}
